/******************************************************************************/
/*
  Purpose:

	QR decomposition of Matrix using cusolver in CUDA
	
  Description:
  
	The program computes a QR factorization of a real m-by-n matrix A as A = Q*R.
	The program does not form the matrix Q explicitly. Instead, Q is represented as a product of min(m, n)
	elementary reflectors :
	Q = H(1)*H(2)* ... *H(k), where k = min(m, n)
	Each H(i) has the form
	H(i) = I - tau*v*vT for real flavors
	where tau is a real scalar stored in tau(i), and v is a real vector with v(1:i-1) = 0 and
	v(i) = 1.
	
	On exit, v(i+1:m) is stored in a(i+1:m, i).
	
	on exit, the elements on and above the diagonal of the array a contain the
	min(n,m)-by-n upper trapezoidal matrix R (R is upper triangular if m ≥ n);

  Modified:

    14 April 2015

  Author:

    Parth Shah
	Email: parthdshah@ufl.edu
*/


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <time.h>
#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>

double timerval () 
{
    struct timeval st;
    gettimeofday(&st, NULL);
    return (st.tv_sec+st.tv_usec*1e-6);
}

// Error Checker for CuSOLVER functions
void cusolveSafeCall(hipsolverStatus_t error1)
{
	if(HIPSOLVER_STATUS_SUCCESS != error1) 
	{
		FILE *fp;
		fp = fopen("results_cuda.txt","a");
		fprintf(fp,"CUDA ERROR in the Cholesky function call or in the buffer and the error is ");
		if(error1 == HIPSOLVER_STATUS_NOT_INITIALIZED) 
			fprintf(fp, "CUSOLVER_STATUS_NOT_INITIALIZED\n");
		if(error1 == HIPSOLVER_STATUS_INVALID_VALUE) 
			fprintf(fp, "CUSOLVER_STATUS_INVALID_VALUE\n");	
		if(error1 == HIPSOLVER_STATUS_ARCH_MISMATCH) 
			fprintf(fp, "CUSOLVER_STATUS_ARCH_MISMATCH\n");
		if(error1 == HIPSOLVER_STATUS_INTERNAL_ERROR) 
			fprintf(fp, "CUSOLVER_STATUS_INTERNAL_ERROR\n");
		fclose(fp);
		exit(EXIT_FAILURE);
	}
}

// Error checker for CUDA memory related operations
void check_error(hipError_t message1)
{
	if( hipSuccess != message1)
	{
		FILE *fp;
		fp = fopen("results_cuda.txt","a");
		fprintf(fp,"CUDA ERROR: %s\n",hipGetErrorString(message1));
		fclose(fp);
		exit(EXIT_FAILURE);
	}
	/*else
	{
		FILE *fp;
		fp = fopen("results_cuda.txt","a");
		//fprintf(fp,"check_error: %s\n",cudaGetErrorString(message1));
		fclose(fp);
	}*/	
}

int main()
{	
	double *A, *tau;
	int m, n, lda;
	
	int i, j, k;
	double avg_time = 0, s_time, e_time;
	
	FILE *fp;							//output file pointer
	fp = fopen("results_cuda.txt","w+");   
	fprintf(fp,"Start:\n");
	fclose(fp);
	
	int *devInfo = NULL; // info in gpu (device copy) 
	int lwork = 0;
	int info_gpu= 0;
	
	check_error(hipMalloc(&devInfo, sizeof(int)));
	
	m = 2;
		
	for (i = 1; i < 16; i++)
	{
		double *d_A = NULL; // linear memory of GPU 
		double *d_tau = NULL; // linear memory of GPU 	
		double *d_work = NULL;

		m *= 2; 						// increase the dimension of Matrix with every iteration
		n = m;			   				// Assuming a square matrix.
		lda = m;		   				// lda: leading dimension of Matrix
		
		A = (double *)calloc(m*n,sizeof(double)); //allocate memory in host
		tau = (double *)calloc(m,sizeof(double));			
		
		//allocate memory in GPU
		check_error(hipMalloc (&d_A , sizeof(double) * lda * m));	
		check_error(hipMalloc (&d_tau, sizeof(double) * m)); 			
		
		hipsolverHandle_t handle;
		hipsolverDnCreate(&handle);
		
		//call function for calculating the work buffer size
		cusolveSafeCall(hipsolverDnDgeqrf_bufferSize( handle, m, n, d_A, lda, &lwork)); 
		
		//allocate the work buffer memory
		check_error(hipMalloc(&d_work, sizeof(double)*lwork));						
		
		avg_time = 0;
			// initialize the matrix
		for(j = 0; j < n; j++)
			for(k = 0; k < m; k++)
				A[k + j * m] = (k + j + 1);
			
		//copy the matrix to GPU
		check_error(hipMemcpy(d_A, A, sizeof(double) * lda * m , hipMemcpyHostToDevice));
		
		for (j = 0; j < 1; j++)
		{	
			info_gpu = 0;
		
			//library function for double precision QR decomposition for a general matrix
			s_time = timerval();
			cusolveSafeCall(hipsolverDnDgeqrf( handle, m, n, d_A, lda, d_tau, d_work, lwork, devInfo));  
			hipDeviceSynchronize(); 
			
			e_time = timerval();
			
			avg_time += (e_time - s_time);
			
			check_error(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost)); 			
			
			if (info_gpu != 0)// if info = 0 the execution is successful else the value in info is illegal element in Matrix
				return info_gpu;				
				
		}
		
		avg_time = avg_time / 1000;
		
		fp = fopen("results_cuda.txt", "a");
		fprintf (fp, "Input size: %d ,Time: %f\n", m, avg_time);  //print the results into the output file
		fclose(fp);
		
		//deallocate memory
		check_error(hipFree(d_A)); 
		check_error(hipFree(d_tau));				
		check_error(hipFree(d_work));
		hipsolverDnDestroy(handle);
		
		free(A);
		free(tau);
	}
	
	check_error(hipFree(devInfo));		

    return 0;
}
