// cholesky decomposition using the library - cuSolver

//**********************************
// cuSolver library must be installed
//**********************************

// link to the documentation of the library - http://docs.nvidia.com/cuda/cusolver/index.html#axzz3V3SakC7i

// author	: yathindra kota 
// mail		: yatkota@ufl.edu
// last modified: 2 April, 2015

#include <stdio.h>
#include <time.h>
#include <string.h>
#include <math.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cudense.h>

#define NUM_THREADS (num_rows*num_cols)
#define BLOCK_WIDTH 1000

void check_error(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err)
	{
		printf("CUDA ERROR: %s (%s).\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}


int main(int argc, char* argv[])
{ 
	cudsHandle_t cudenseH = NULL;
    	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;  
	
	int lwork = 0;
	int *devInfo = NULL;
	int info_gpu_h = 0;
	int num_rows = 4;
	int num_cols = 4;
	int num_iterations = 10;
	
	FILE *fp;
	
	// create cudense/cublas handle
    	cusolver_status = cudsCreate(&cudenseH);
    	hipError_t cudaStat1;
	float time_iterations = 0; //time for given set of iterations
	
	for(int matsize = 0; matsize < 10; matsize++)
	{	
		double *h_matrix;
		double *h_matrix_ouput;
		
		h_matrix = malloc(num_rows*num_cols, sizeof(double));
		h_matrix_ouput = malloc(num_rows*num_cols, sizeof(double));
		
		int temp;
		for(temp = 0; temp < num_iterations; temp++)
		{
			//initialize the matrix to a random set of values (0 to 10) of type float 	
			for(int tempi=0; tempi<num_rows;tempi++)
			{
				for(int tempj=0; tempi<num_cols;tempj++)	
				{
					h_matrix[(tempi*num_rows) + tempj] = (rand()%100)/10; // set the input matrix elements 
				}	
			}		

			// allocate memory in the GPU and also copy the input matrix from hot to device
			double *d_matrix_input;
			check_error(hipMalloc((void **) &d_matrix_input, num_rows * num_cols * sizeof(float)));
			check_error(hipMemcpy(*d_matrix_input, *h_matrix, sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice));

			//------	
			// alocation of the buffer in the GPU
			// The following function allocates required amount of memory in the GPU and this
			// function is specific to Cholesky function which is called next
			// This function is also from the library cuSolver
				
			hipsolverStatus_t hipsolverDnDpotrf_bufferSize(cudenseH,
						 HIPBLAS_FILL_MODE_LOWER, /*hipblasFillMode_t uplo, Maybe "HIPBLAS_FILL_MODE_LOWER" */
						 num_rows,
						 d_matrix_input,
						 num_rows,
						 &Lwork);
			
			double *d_work = NULL; 	
			cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*Lwork);
			
			cudaStat1 = hipDeviceSynchronize(); // synchronization
			
			timer.start();
			
			// implementation of Cholesky
			hipsolverStatus_t hipsolverDnDpotrf(cudenseH,
				   HIPBLAS_FILL_MODE_LOWER,/* not sure, needs to be checked*/
				   num_rows,
				   d_matrix_input,
				   num_rows,
				   d_work, /*not sure */
				   Lwork,
				   devInfo );
			
			cudaStat1 = hipDeviceSynchronize(); // synchronization
			
			timer.stop();
			
			time_iterations += timer.Elapsed;	
			
			//// check if Cholesky is good or not
			cudaStat1 = hipMemcpy(&info_gpu_h, devInfo, sizeof(int), hipMemcpyDeviceToHost);
			
			if(info_gpu_h != 0)
			{
				fp = fopen("cholesky_output.txt","w+");
				fprintf(fp,"Unsuccessful execution. DevInfo is not zero. Iterations number = %d \n", temp);
				fclose(fp);
			}	
		}
		cudaStat1 = hipMemcpy(h_matrix_ouput, d_work, sizeof(double)*num_rows*num_cols, hipMemcpyDeviceToHost); 
		//d_work needs to checked
		
		fp = fopen("cholesky_output.txt","a");
		fprintf(fp,"Time elapsed(average for the specified iterations) = %g ms, number of rows is %d\n", time_iterations/num_iterations,num_rows);
		fclose(fp);
		
		time_iterations = 0;
		
		free(h_matrix);
		free(h_matrix_ouput);		
			
		num_cols *= 2;
		num_rows *= 2;
	}
	// print output to file if required
	/*
	
	fprintf(fp,"output is\n");	
	for(tempi=0; tempi<num_rows;tempi++)
	{
		fprintf(fp,"\n");
		for(tempj=0; tempi<num_cols;tempj++)	
		{
			fp = fopen("cholesky_output.txt","a");
			fprintf(fp,"%f", h_matrix_ouput[(tempj * num_rows) + tempi]);
			fclose(fp);
		}		
	}
	
	*/
	//de-allocating the memory
	
	if(d_matrix_input)
		hipFree(d_matrix_input);
		
	if(d_work)
		hipFree(d_work);
	
    	if (cudenseH) 
		cudsDestroy(cudenseH); 	

	return 0;	
}
